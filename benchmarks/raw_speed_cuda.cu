#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

#include "phillox.h"
#include "threefry.h"
#include "squares.h"
#include "tyche.h"

using std::cout;
using std::endl;

const int N = 268435456; // no of 32 bits integers required for 1 GB data

template<typename RNG>
__global__ void measure_speed_cuda_kernel(uint32_t *global_sum_dev, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    uint32_t sum = 0;

    // Assuming the RNG class has been defined to work in device code
    RNG rng(12345 + idx, 0);

    for (int i = idx; i < N; i += stride) {
        sum += rng.template draw<uint32_t>();
    }

    // Atomic add to the global sum
    atomicAdd(global_sum_dev, sum);
}


template<typename RNG>
double measure_speed_cuda() {
    using namespace std::chrono;
    uint32_t global_sum = 0;
    uint32_t *global_sum_dev;

    // Allocate memory for global_sum on the device
    hipMalloc((void **)&global_sum_dev, sizeof(uint32_t));
    hipMemcpy(global_sum_dev, &global_sum, sizeof(uint32_t), hipMemcpyHostToDevice);

    auto start = high_resolution_clock::now();

    // Launch the CUDA Kernel
    int numBlocks = 256;
    int numThreadsPerBlock = 256;
    measure_speed_cuda_kernel<RNG><<<numBlocks, numThreadsPerBlock>>>(global_sum_dev, N);

    hipDeviceSynchronize();

    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);

    hipMemcpy(&global_sum, global_sum_dev, sizeof(uint32_t), hipMemcpyDeviceToHost);

    hipFree(global_sum_dev);

    global_sum &= 1;  // avoid polluting the output

    // Total gigabytes produced
    double total_gb = N * sizeof(uint32_t) / 1e9;
    double time_taken = duration.count() / 1e6;

    // Speed: GB/s
    double speed = total_gb / time_taken;

    std::cout << "Speed: " << speed << " GB/s " << global_sum << std::endl;
    return duration.count();
}

int main(){

    cout<<"Phillox: "<<endl;
    measure_speed_cuda<Phillox>();

    cout<<"Threefry: "<<endl;
    measure_speed_cuda<Threefry>();

    cout<<"Squares: "<<endl;
    measure_speed_cuda<Squares>();

    cout<<"Tyche: "<<endl;
    measure_speed_cuda<Tyche>();

    return 0;
}