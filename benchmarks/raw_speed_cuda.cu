#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

#include "phillox.h"
#include "threefry.h"
#include "squares.h"
#include "tyche.h"

using std::cout;
using std::endl;

const int N = 268435456; // no of 32 bits integers required for 1 GB data

template<typename RNG>
__global__ void measure_speed_cuda_kernel(uint32_t *global_sum_dev, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    uint32_t sum = 0;

    // Assuming the RNG class has been defined to work in device code
    RNG rng(12345 + idx, 0);

    for (int i = idx; i < N; i += stride) {
        sum += rng.template draw<uint32_t>();
    }

    // Atomic add to the global sum
    atomicAdd(global_sum_dev, sum);
}


template<typename RNG>
double measure_speed_cuda(int numSMs) {
    using namespace std::chrono;
    uint32_t global_sum = 0;
    uint32_t *global_sum_dev;

    // Allocate memory for global_sum on the device
    hipMalloc((void **)&global_sum_dev, sizeof(uint32_t));
    hipMemcpy(global_sum_dev, &global_sum, sizeof(uint32_t), hipMemcpyHostToDevice);

    auto start = high_resolution_clock::now();

    // Launch the CUDA Kernel
    int numBlocks = numSMs * 4;
    int numThreadsPerBlock = 256;
    int nums_per_thread = N / (numBlocks * numThreadsPerBlock);
    measure_speed_cuda_kernel<RNG><<<numBlocks, numThreadsPerBlock>>> \
        (global_sum_dev, nums_per_thread);

    hipDeviceSynchronize();

    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);

    hipMemcpy(&global_sum, global_sum_dev, sizeof(uint32_t), hipMemcpyDeviceToHost);

    hipFree(global_sum_dev);

    //global_sum &= 1;  // avoid polluting the output

    // Total gigabytes produced
    double total_gb = N * sizeof(uint32_t) / 1e9;
    double time_taken = duration.count() / 1e6;

    // Speed: GB/s
    double speed = total_gb / time_taken;

    std::cout << "Speed: " << speed << " GB/s " << global_sum << std::endl;
    return duration.count();
}

int main(){
    hipDeviceProp_t deviceProp;
    int device;
    hipGetDevice(&device); // Get current device
    hipGetDeviceProperties(&deviceProp, device);

    std::cout << "Number of Streaming Multiprocessors (SMs): " << deviceProp.multiProcessorCount << std::endl;

    cout<<"Threefry: "<<endl;
    measure_speed_cuda<Threefry>(deviceProp.multiProcessorCount);

    cout<<"Phillox: "<<endl;
    measure_speed_cuda<Phillox>(deviceProp.multiProcessorCount);

    cout<<"Squares: "<<endl;
    measure_speed_cuda<Squares>(deviceProp.multiProcessorCount);

    cout<<"Tyche: "<<endl;
    measure_speed_cuda<Tyche>(deviceProp.multiProcessorCount);

    return 0;
}